/*************************************************************************
 * Copyright (c) 2016-2022, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "hip/hip_runtime.h"
#include "common.h"

void AllShiftGetCollByteCount(size_t *sendcount, size_t *recvcount, size_t *paramcount, size_t *sendInplaceOffset, size_t *recvInplaceOffset, size_t count, int nranks) {
  *sendcount = count;
  *recvcount = count;
  *sendInplaceOffset = 0;
  *recvInplaceOffset = 0;
  *paramcount = *sendcount;
}


testResult_t AllShiftInitData(struct threadArgs* args, ncclDataType_t type, ncclRedOp_t op, int root, int rep, int in_place) {
  size_t sendcount = args->sendBytes / wordSize(type);
  size_t recvcount = args->expectedBytes / wordSize(type);
  int nranks = args->nProcs*args->nThreads*args->nGpus;

  for (int i=0; i<args->nGpus; i++) {
    CUDACHECK(hipSetDevice(args->gpus[i]));
    int rank = ((args->proc*args->nThreads + args->thread)*args->nGpus + i);
    CUDACHECK(hipMemset(args->recvbuffs[i], 0, args->expectedBytes));
    void* data = in_place ? args->recvbuffs[i] : args->sendbuffs[i];
    TESTCHECK(InitData(data, sendcount, 0, type, ncclSum, 33*rep + rank, 1, 0));
    int peer = (rank-1+nranks)%nranks;
    TESTCHECK(InitData(args->expected[i], recvcount, 0, type, ncclSum, 33*rep + peer, 1, 0));
    CUDACHECK(hipDeviceSynchronize());
  }
  // We don't support in-place sendrecv
  args->reportErrors = in_place ? 0 : 1;
  return testSuccess;
}


void AllShiftGetBw(size_t count, int typesize, double sec, double* algBw, double* busBw, int nranks) {
  double baseBw = (double)(count * typesize) / 1.0E9 / sec;

  *algBw = baseBw;
  *busBw = baseBw;
}


testResult_t AllShiftRunColl(void* sendbuff, void* recvbuff, size_t count, ncclDataType_t type, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream) {
  int nRanks;
  NCCLCHECK(ncclCommCount(comm, &nRanks));
  int rank;
  NCCLCHECK(ncclCommUserRank(comm, &rank));
  int recvPeer = (rank-1+nRanks) % nRanks;
  int sendPeer = (rank+1) % nRanks;

  NCCLCHECK(ncclGroupStart());
  NCCLCHECK(ncclSend(sendbuff, count, type, sendPeer, comm, stream));
  NCCLCHECK(ncclRecv(recvbuff, count, type, recvPeer, comm, stream));
  NCCLCHECK(ncclGroupEnd());
  return testSuccess;
}



struct testColl allShiftTest = {
  "AllShift",
  AllShiftGetCollByteCount,
  AllShiftInitData,
  AllShiftGetBw,
  AllShiftRunColl
};


void AllShiftGetBuffSize(size_t *sendcount, size_t *recvcount, size_t count, int nranks) {
  size_t paramcount, sendInplaceOffset, recvInplaceOffset;
  AllShiftGetCollByteCount(sendcount, recvcount, &paramcount, &sendInplaceOffset, &recvInplaceOffset, count, nranks);
}


testResult_t AllShiftRunTest(struct threadArgs* args, int root, ncclDataType_t type, const char* typeName, ncclRedOp_t op, const char* opName) {
  args->collTest = &allShiftTest;
  ncclDataType_t *run_types;
  const char **run_typenames;
  int type_count;

  if ((int)type != -1) {
    type_count = 1;
    run_types = &type;
    run_typenames = &typeName;
  } else {
    type_count = test_typenum;
    run_types = test_types;
    run_typenames = test_typenames;
  }

  for (int i=0; i<type_count; i++) {
    TESTCHECK(TimeTest(args, run_types[i], run_typenames[i], (ncclRedOp_t)0, "none", -1));
  }
  return testSuccess;
}

struct testEngine AllShiftEngine = {
  AllShiftGetBuffSize,
  AllShiftRunTest
};

#pragma weak ncclTestEngine=AllShiftEngine
